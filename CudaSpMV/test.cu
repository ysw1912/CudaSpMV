#include "hip/hip_runtime.h"
#include "test.h"

#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\sort.h>

#if defined(__cplusplus) && defined(__HIPCC__)
# include "cooperative_groups_helpers.h"
namespace cg = cooperative_groups;
#endif

namespace test
{
	// x�Ƿ���2���ݴ�
	static inline bool is_pow_of_2(uint32_t x)
	{
		return !(x & (x - 1));
	}

	// ���ڵ���x����С2���ݴ���
	static inline uint32_t next_pow_of_2(uint32_t x)
	{
		if (is_pow_of_2(x))
			return x;
		x |= x >> 1;
		x |= x >> 2;
		x |= x >> 4;
		x |= x >> 8;
		x |= x >> 16;
		return x + 1;
	}

	// BRC�����һ��
	struct st
	{
		uint32_t a;
		uint32_t b;
		uint32_t c;

		__host__ __device__ bool operator<(const st &rhs) const
		{
			return this->b > rhs.b;
		}
	};

	void test01()
	{
		int N = 100000;
		thrust::host_vector<st> v(N);
		for (int i = 0; i < N; ++i) {
			v[i].a = 0;
			v[i].b = N - i;
			v[i].c = i;
		}
		printf("1\n");
		thrust::device_vector<st> dv(N);
		printf("2\n");
		thrust::copy(v.begin(), v.end(), dv.begin());
		printf("3\n");
		thrust::stable_sort(dv.begin(), dv.end());
		printf("4\n");
		v = dv;
		printf("5\n");
		for (int i = 0; i < 7; i++) {
			printf("(%d - %d - %d), ", v[i].a, v[i].b, v[i].c);
		}
		printf("\n");
	}

	extern "C" const uint32_t MAX_BATCH_ELEMENTS = 64 * 1048576;	// 2^6 * 2^20
#define BLOCK_SIZE 1024

	void test02()
	{
		uint32_t *d_Input, *d_Output;
		uint32_t *h_Input, *h_OutputCPU, *h_OutputGPU;
		uint32_t N;
		scanf("%d", &N);
		uint32_t d_N = next_pow_of_2(N);
		if (d_N < 4 * BLOCK_SIZE)
			d_N = 4 * BLOCK_SIZE;

		h_Input = (uint32_t*)malloc(d_N * sizeof(uint32_t));
		h_OutputCPU = (uint32_t*)malloc(N * sizeof(uint32_t));
		h_OutputGPU = (uint32_t*)malloc(N * sizeof(uint32_t));
		for (uint32_t i = 0; i < N; i++)
			h_Input[i] = 1;
		for (uint32_t i = N; i < d_N; i++)
			h_Input[i] = 0;

		checkCudaError(hipMalloc((void**)&d_Input, d_N * sizeof(uint32_t)));
		checkCudaError(hipMalloc((void**)&d_Output, d_N * sizeof(uint32_t)));
		checkCudaError(hipMemcpy(d_Input, h_Input, d_N * sizeof(uint32_t), hipMemcpyHostToDevice));
	
		ExclusiveScanHost(h_OutputCPU, h_Input, N);
		printf("[CPU] "); Print<uint32_t>(h_OutputCPU, N, false);

		printf("Running scan...\n[CPU arrayLength: %u, GPU arrayLength: %u]\n", N, d_N);
		checkCudaError(hipDeviceSynchronize());
		ExclusiveScan(d_Output, d_Input, d_N);
		checkCudaError(hipDeviceSynchronize());

		// ֻ����N��Ԫ��
		checkCudaError(hipMemcpy(h_OutputGPU, d_Output, N * sizeof(uint32_t), hipMemcpyDeviceToHost));
		printf("[GPU] "); Print<uint32_t>(h_OutputGPU, N, false);

		checkCudaError(hipFree(d_Input));
		checkCudaError(hipFree(d_Output));

		printf("Validating the results...\n");
		int flag = 1;
		for (uint32_t i = 0; i < N; i++) {
			if (h_OutputCPU[i] != h_OutputGPU[i]) {
				flag = 0;
				break;
			}
		}
		printf(" ...Results %s\n\n", (flag == 1) ? "Match" : "DON'T Match !!!");
	}

	/*
	** return ����factor
	** factor * 2^log2L = L
	** ��L=5, ��log2L=0, factor=5, ��Ϊ5 * 2^0 = 5
	** ��L=6, ��log2L=1, factor=3, ��Ϊ3 * 2^1 = 6
	**/
	static uint32_t factorRadix2(uint32_t &log2L, uint32_t L)
	{
		if (!L) {
			log2L = 0;
			return 0;
		}
		else {
			for (log2L = 0; (L & 1) == 0; L >>= 1, log2L++);
			return L;
		}
	}

	/*
	** return �������
	** ������������ȡ��
	** �� 5 / 2 ���� 3
	**/
	static uint32_t iDivUp(uint32_t dividend, uint32_t divisor)
	{
		return ((dividend % divisor) == 0) ? (dividend / divisor) : (dividend / divisor + 1);
	}

	void ExclusiveScanHost(uint32_t* dst, uint32_t* src, uint32_t size)
	{
		dst[0] = 0;
		for (uint32_t i = 1; i < size; i++)
			dst[i] = src[i - 1] + dst[i - 1];			
	}

	inline __device__
	uint32_t scan1Inclusive(uint32_t idata, volatile uint32_t *s_Data, uint32_t size, cg::thread_block cta)
	{
		uint32_t pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
		s_Data[pos] = 0;
		pos += size;
		s_Data[pos] = idata;
#pragma unroll
		for (uint32_t offset = 1; offset < size; offset <<= 1) {
			cg::sync(cta);
			uint32_t t = s_Data[pos] + s_Data[pos - offset];
			cg::sync(cta);
			s_Data[pos] = t;
		}

		return s_Data[pos];
	}

	inline __device__
	uint32_t scan1Exclusive(uint32_t idata, volatile uint32_t *s_Data, uint32_t size, cg::thread_block cta)
	{
		return scan1Inclusive(idata, s_Data, size, cta) - idata;
	}


	inline __device__
	uint4 scan4Inclusive(uint4 idata4, volatile uint32_t *s_Data, uint32_t size, cg::thread_block cta)
	{
		//Level-0 inclusive scan
		idata4.y += idata4.x;
		idata4.z += idata4.y;
		idata4.w += idata4.z;

		//Level-1 exclusive scan
		uint32_t oval = scan1Exclusive(idata4.w, s_Data, size, cta);
		//printf("%d %d\n", idata4.w, oval);

		idata4.x += oval;
		idata4.y += oval;
		idata4.z += oval;
		idata4.w += oval;
		
		return idata4;
	}

	inline __device__
	uint4 scan4Exclusive(uint4 idata4, volatile uint32_t *s_Data, uint32_t size, cg::thread_block cta)
	{
		uint4 odata4 = scan4Inclusive(idata4, s_Data, size, cta);
		odata4.x -= idata4.x;
		odata4.y -= idata4.y;
		odata4.z -= idata4.z;
		odata4.w -= idata4.w;
		return odata4;
	}

	__global__ void ExclusiveScanShared(uint4 *d_Dst, uint4 *d_Src, uint32_t size)
	{
		cg::thread_block cta = cg::this_thread_block();
		//__shared__ uint32_t s_Data[2 * BLOCK_SIZE];
		uint32_t* s_Data = SharedMemory<uint32_t>();

		uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;
		uint4 idata4 = d_Src[pos];
		uint4 odata4 = scan4Exclusive(idata4, s_Data, size, cta);
		d_Dst[pos] = odata4;
	}

	// �� list[ÿ(4 * BLOCK_SIZE)��Ԫ�صĺ�] ��exclusive scan
	__global__ void ExclusiveScanShared2(uint32_t* d_Buf, uint32_t* d_Dst, uint32_t* d_Src, uint32_t N, uint32_t size)
	{
		cg::thread_block cta = cg::this_thread_block();
		__shared__ uint32_t s_Data[2 * BLOCK_SIZE];

		uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;

		uint32_t idata = 0;
		if (pos < N)	// ������д���һ��block�ķǻ�Ծ�߳�
			// ��d_Dst��ȡÿ(4 * BLOCK_SIZE)�����һ��Ԫ�أ���Ϊexclusive scan�������
			// ����d_Src��ӦԪ�صõ�inclusive scan�Ľ��������(4 * BLOCK_SIZE)��Ԫ�صĺ�
			idata = d_Dst[(4 * BLOCK_SIZE) - 1 + (4 * BLOCK_SIZE) * pos]
				  + d_Src[(4 * BLOCK_SIZE) - 1 + (4 * BLOCK_SIZE) * pos];

		// ��idata��exclusive scan
		uint32_t odata = scan1Exclusive(idata, s_Data, size, cta);

		if (pos < N) {
			d_Buf[pos] = odata;
			//printf("idata = %d, odata = %d\n", idata, odata);
		}
	}

	// ��d_Buf��ÿ��block�Ľ������d_Dst
	__global__ void UniformUpdate(uint4 *d_Dst, uint32_t *d_Buf)
	{
		cg::thread_block cta = cg::this_thread_block();
		__shared__ uint32_t buf;
		uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadIdx.x == 0)
			buf = d_Buf[blockIdx.x];
		cg::sync(cta);

		uint4 data4 = d_Dst[pos];
		data4.x += buf;
		data4.y += buf;
		data4.z += buf;
		data4.w += buf;
		d_Dst[pos] = data4;
	}

	void ExclusiveScanShort(uint32_t *d_Dst, uint32_t *d_Src, uint32_t size)
	{
		// ����Ƿ�����block��������
		assert(size % (4 * BLOCK_SIZE) == 0);

		ExclusiveScanShared<<<size / (4 * BLOCK_SIZE), BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t) >>>((uint4*)d_Dst, (uint4*)d_Src, size / 4);
	}

	void ExclusiveScanLarge(uint32_t *d_Dst, uint32_t *d_Src, uint32_t size)
	{
		// һ��block����(4 * BLOCK_SIZE)��Ԫ�ص�exclusive scan
		ExclusiveScanShared<<<size / (4 * BLOCK_SIZE), BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>>((uint4*)d_Dst, (uint4*)d_Src, BLOCK_SIZE);

		uint32_t *d_Buf;	// ���ExclusiveScanShared2�Ľ��
		checkCudaError(hipMalloc((void**)&d_Buf, (MAX_BATCH_ELEMENTS / (4 * BLOCK_SIZE)) * sizeof(uint32_t)));

		const uint32_t blockCount2 = iDivUp(size / (4 * BLOCK_SIZE), BLOCK_SIZE);
		ExclusiveScanShared2<<<blockCount2, BLOCK_SIZE>>>(d_Buf, d_Dst, d_Src, size / (4 * BLOCK_SIZE), size / (4 * BLOCK_SIZE));

		UniformUpdate<<<size / (4 * BLOCK_SIZE), BLOCK_SIZE>>>((uint4*)d_Dst, d_Buf);
		
		checkCudaError(hipFree(d_Buf));
	}

	void ExclusiveScan(uint32_t* d_Dst, uint32_t* d_Src, uint32_t size)
	{
		// ���size�Ƿ���2���ݴ�
		assert(is_pow_of_2(size));
		//uint32_t log2L;
		//assert(factorRadix2(log2L, size) == 1);

		if (size <= 4 * BLOCK_SIZE)
			ExclusiveScanShort(d_Dst, d_Src, size);
		else
			ExclusiveScanLarge(d_Dst, d_Src, size);
	}
}