#include "spmv.h"

// CuSparse_SpMV��װ��float�����أ�
void spmvCuSparse(hipsparseHandle_t &handle, int m, int n, int nnz, const float *alpha,
	hipsparseMatDescr_t &descr, const float *value, const int *rowPtr, const int *col,
	const float *x, const float *beta, float *y) {
	checkCuSparseError(hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz, alpha, descr, value, rowPtr, col, x, beta, y));
}
// CuSparse_SpMV��װ��double�����أ�
void spmvCuSparse(hipsparseHandle_t &handle, int m, int n, int nnz, const double *alpha,
	hipsparseMatDescr_t &descr, const double *value, const int *rowPtr, const int *col,
	const double *x, const double *beta, double *y) {
	checkCuSparseError(hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz, alpha, descr, value, rowPtr, col, x, beta, y));
}

